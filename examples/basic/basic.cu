#include "hip/hip_runtime.h"
#include "../common.h"

#include <hip/hip_runtime.h>

#include <cutangent/cutangent.cuh>
#include <cutangent/format.h>

#include <iostream>

using cu::tangent;

constexpr auto f(auto x, auto y)
{
    auto print = [](auto x) { printf("{%g, %g}\n", x.v, x.d); };

    auto a  = x + y;
    auto b  = x - y;
    auto c  = x * y;
    auto d  = x / y;
    auto e  = max(x, y);
    auto f  = min(x, y);
    auto g  = mid(x, y, y);
    auto h  = sin(x);
    auto i  = cos(x);
    auto j  = exp(x);
    auto k  = log(x);
    auto l  = pown(x, 2);
    auto m  = x * 2;
    auto n  = log2(x);
    auto o  = log10(x);
    auto p  = tan(x);
    auto q  = asin(x);
    auto r  = acos(x);
    auto s  = atan(x);
    auto t  = sinh(x);
    auto u  = cosh(x);
    auto v  = tanh(x);
    auto w  = asinh(x);
    auto aa = acosh(x);
    auto bb = atanh(x);
    auto cc = atan2(y, x);
    auto dd = atan2(y, 2.0);
    auto ee = atan2(2.0, x);

    print(a);
    print(b);
    print(c);
    print(d);
    print(e);
    print(f);
    print(g);
    print(h);
    print(i);
    print(j);
    print(k);
    print(l);
    print(m);
    print(n);
    print(o);
    print(p);
    print(q);
    print(r);
    print(s);
    print(t);
    print(u);
    print(v);
    print(w);
    print(aa);
    print(bb);
    print(cc);
    print(dd);
    print(ee);
    return a;
}

__global__ void kernel(tangent<double> *xs, tangent<double> *ys,
                       tangent<double> *res, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        res[i] = f(xs[i], ys[i]);
    }
}

int main()
{
    constexpr int n = 16;
    using T         = tangent<double>;
    T xs[n], ys[n], res[n];

    // generate dummy data
    for (int i = 0; i < n; i++) {
        double v = i;
        xs[i]    = { v, 1.0 };
        ys[i]    = { v, 0.0 };
    }

    // for (auto el : xs) {
    //     std::cout << el << std::endl;
    // }

    T *d_xs, *d_ys, *d_res;
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_ys, n * sizeof(*ys)));
    CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));

    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ys, ys, n * sizeof(*ys), hipMemcpyHostToDevice));

    kernel<<<n, 1>>>(d_xs, d_ys, d_res, n);

    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    // for (auto el : res) {
    //     std::cout << el << std::endl;
    // }

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_ys));
    CUDA_CHECK(hipFree(d_res));

    return 0;
}
