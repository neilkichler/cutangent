#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cumccormick/cumccormick.cuh>

#include <cutangent/cutangent.cuh>

template<typename T>
using mc = cu::mccormick<T>;

int main()
{
    // constexpr int n = 256;
    // using T = mc<double>;
    // T xs[n], ys[n], res[n];
    //
    // // generate dummy data
    // for (int i = 0; i < n; i++) {
    //     double v = i;
    //     xs[i] = { .cv = -v, .cc = v, .box = { .lb = -v, .ub = v } };
    //     ys[i] = { .cv = -v, .cc = v, .box = { .lb = -v, .ub = v } };
    // }
    //
    // mc<double> *d_xs, *d_ys, *d_res;
    // CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    // CUDA_CHECK(hipMalloc(&d_ys, n * sizeof(*ys)));
    // CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));
    //
    // CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMemcpy(d_ys, ys, n * sizeof(*ys), hipMemcpyHostToDevice));
    //
    // kernel<<<n, 1>>>(d_xs, d_ys, d_res, n);
    //
    // CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));
    //
    // auto r = res[0];
    // printf("beale(0, 0) = " MCCORMICK_FORMAT "\n", r.box.lb, r.cv, r.cc, r.box.ub);
    //
    // CUDA_CHECK(hipFree(d_xs));
    // CUDA_CHECK(hipFree(d_ys));
    // CUDA_CHECK(hipFree(d_res));

    return 0;
}
