#include "hip/hip_runtime.h"
#include "../common.h"

#include <hip/hip_runtime.h>

#include <cumccormick/cumccormick.cuh>
#include <cumccormick/format.h>

#include <cutangent/cutangent.cuh>
#include <cutangent/format.h>

#include <iostream>

namespace blackscholes
{

template<typename T>
struct parameters
{
    T r;     // interest rate
    T S0;    // spot price
    T tau;   // time until maturity
    T K;     // strike price
    T sigma; // std. dev. of stock return (i.e., volatility)
};

// Call price given the Black-Scholes model
template<typename T>
constexpr auto call(parameters<T> params)
{
    auto [r, S0, tau, K, sigma] = params;
    assert((S0 > 0.0) && (tau > 0.0) && (sigma > 0.0) && (K > 0.0));

    using std::exp;
    using std::log;
    using std::pow;
    using std::sqrt;

    auto normcdf = [](auto x) {
        using std::erfc;
        return 0.5 * erfc(-x * M_SQRT1_2);
    };

    auto discount_factor = exp(-r * tau);
    auto variance        = sigma * sqrt(tau);
    auto forward_price   = S0 / discount_factor;

    auto dp         = (log(forward_price / K) + 0.5 * pow(sigma, 2) * tau) / variance;
    auto dm         = dp - variance;
    auto call_price = discount_factor * (forward_price * normcdf(dp) - K * normcdf(dm));
    return call_price;
}

// Derivative of call price w.r.t. S0 (spot price)
template<typename T>
constexpr auto delta(parameters<T> params)
{
    auto [r, S0, tau, K, sigma] = params;
    assert((S0 > 0.0) && (tau > 0.0) && (sigma > 0.0) && (K > 0.0));

    using std::exp;
    using std::log;
    using std::pow;
    using std::sqrt;

    auto normcdf = [](auto x) {
        using std::erfc;
        return 0.5 * erfc(-x * 1.0 / std::numbers::sqrt2);
    };

    auto discount_factor = exp(-r * tau);
    auto variance        = sigma * sqrt(tau);
    auto forward_price   = S0 / discount_factor;

    auto dp = (log(forward_price / K) + 0.5 * pow(sigma, 2) * tau) / variance;
    return normcdf(dp);
}

// Derivative of call price w.r.t. sigma (i.e., volatility)
template<typename T>
constexpr auto vega(parameters<T> params)
{
    auto [r, S0, tau, K, sigma] = params;
    assert((S0 > 0.0) && (tau > 0.0) && (sigma > 0.0) && (K > 0.0));

    using std::exp;
    using std::log;
    using std::pow;
    using std::sqrt;

    auto normpdf = [](auto x) {
        return exp(-pow(x, 2) / 2.0) / sqrt(2.0 * std::numbers::pi);
    };

    auto discount_factor = exp(-r * tau);
    auto variance        = sigma * sqrt(tau);
    auto forward_price   = S0 / discount_factor;

    auto dp = (log(forward_price / K) + 0.5 * pow(sigma, 2) * tau) / variance;

    return S0 * normpdf(dp) * sqrt(tau);
}

}; // namespace blackscholes

__global__ void bs_kernel(auto *ps, auto *res, std::integral auto n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        res[i] = blackscholes::call(ps[i]);
    }
}

int main()
{
    constexpr int n = 1;

    using T = cu::tangent<cu::mccormick<double>>;
    blackscholes::parameters<T> xs[n]{};
    T res[n];

    // generate dummy data
    for (int i = 0; i < n; i++) {
        // double v = i + 1;

        value(xs[i].r)  = 0.01;
        value(xs[i].S0) = { .cv = 99.5, .cc = 100.5, .box = { .lb = 99.5, .ub = 100.5 } };
        // value(xs[i].tau)   = 0.01 * v;
        value(xs[i].tau)   = 3.0 / 12.0;
        value(xs[i].K)     = 95.0;
        value(xs[i].sigma) = 0.5;

        // update seeds to compute derivative w.r.t S0
        derivative(xs[i].S0) = 1.0;
    }

    std::cout << "---- Computing Delta ----" << std::endl;
    std::cout << "S0: " << xs[0].S0 << std::endl;
    std::cout << "sigma: " << xs[0].sigma << std::endl;

    blackscholes::parameters<T> *d_xs;
    T *d_res;
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));
    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    bs_kernel<<<n, 1>>>(d_xs, d_res, n);
    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    auto delta = res[0];
    std::cout << "Black Scholes w.r.t. S0 (i.e., Delta): " << delta << std::endl;

    blackscholes::parameters<double> params;
    params = { 0.01, 99.5, 3.0 / 12.0, 95.0, 0.5 };
    std::cout << "Analytic Delta(S0= 99.5): " << blackscholes::delta(params) << std::endl;
    params = { 0.01, 100.0, 3.0 / 12.0, 95.0, 0.5 };
    std::cout << "Analytic Delta(S0=100.0): " << blackscholes::delta(params) << std::endl;
    params = { 0.01, 100.5, 3.0 / 12.0, 95.0, 0.5 };
    std::cout << "Analytic Delta(S0=100.5): " << blackscholes::delta(params) << std::endl;

    // update seeds to compute derivative w.r.t sigma
    for (int i = 0; i < n; i++) {
        derivative(xs[i].sigma) = 1.0;
        derivative(xs[i].S0)    = 0.0;
    }

    std::cout << "---- Computing Vega ----" << std::endl;
    std::cout << "S0: " << xs[0].S0 << std::endl;
    std::cout << "sigma: " << xs[0].sigma << std::endl;

    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    bs_kernel<<<n, 1>>>(d_xs, d_res, n);
    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    auto vega = res[0];
    std::cout << "Black Scholes w.r.t. sigma (i.e., Vega): " << vega << std::endl;

    params = { 0.01, 99.5, 3.0 / 12.0, 95.0, 0.5 };
    std::cout << "Analytic Vega(S0= 99.5): " << blackscholes::vega(params) << std::endl;
    params = { 0.01, 100.0, 3.0 / 12.0, 95.0, 0.5 };
    std::cout << "Analytic Vega(S0=100.0): " << blackscholes::vega(params) << std::endl;
    params = { 0.01, 100.5, 3.0 / 12.0, 95.0, 0.5 };
    std::cout << "Analytic Vega(S0=100.5): " << blackscholes::vega(params) << std::endl;

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_res));

    return 0;
}
