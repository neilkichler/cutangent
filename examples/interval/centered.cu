#include "hip/hip_runtime.h"
#include "../common.h"

#include <hip/hip_runtime.h>

#include <cuinterval/cuinterval.h>
#include <cuinterval/format.h>

#include <cutangent/cutangent.cuh>

#include <iostream>

using cu::tangent;

using I = cu::interval<double>;

using T = tangent<I>;

constexpr auto f(auto x)
{
    using std::pow;

    return 3.0 * pow(x, 3) + pow(x, 2) - 5.0 * x - 1.0;
}

__global__ void centered_form(T *xs, I *res, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        auto r = f(xs[i]);

        auto L = derivative(r);
        auto X = value(xs[i]);
        auto c = mid(value(xs[i]));

        auto centered_form = f(c) + L * (X - c);

        res[i] = centered_form;
    }
}

int main()
{
    constexpr int n = 1;
    T xs[n];
    I res[n];

    value(xs[0])      = { -1.0, 1.0 };
    derivative(xs[0]) = { 1.0, 1.0 };

    T *d_xs;
    I *d_res;
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));

    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));

    centered_form<<<n, 1>>>(d_xs, d_res, n);

    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    auto r = res[0];
    std::cout << "f(x) = x^3 + x^2 - 5x - 1" << std::endl;
    std::cout << "X = " << value(xs[0]) << std::endl;
    std::cout << "centered form for f at X: " << r << std::endl;

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_res));

    return 0;
}
