#include "hip/hip_runtime.h"
#include "../common.h"

#include <hip/hip_runtime.h>

#include <cuinterval/cuinterval.h>
#include <cuinterval/format.h>

#include <cutangent/cutangent.cuh>
#include <cutangent/format.h>

#include <iostream>

using cu::tangent;

template<typename T>
using I = cu::interval<T>;

constexpr auto f(auto x, auto y)
{
    // Currently supported functions:

    // auto a = neg(x);
    // auto a = add(x, y);
    // auto a = sub(x, y);
    // auto a = mul(x, y);
    // auto a = div(x, y);
    // auto a = x + y;
    // auto a = x - y;
    // auto a = x / y;
    // auto a = x * y;
    // auto a = sqr(x);
    // auto a = sqrt(x);
    // auto a = abs(x);
    // auto a = exp(x);
    // auto a = log(x);
    // auto a = recip(x);
    // auto a = cos(x);
    // auto a = pown(x, 3);
    // auto a = pown(x, 4.0);
    // auto a = pow(x, 4.0);
    auto a = pow(x, y);
    // auto a = max(x, y);
    // auto a = min(x, y);
    // auto a = hull(x, y);
    return a;
}

__global__ void kernel(I<tangent<double>> *xs, I<tangent<double>> *ys, I<tangent<double>> *res, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        res[i] = f(xs[i], ys[i]);
    }
}

int main()
{
    constexpr int n = 1;
    using T         = I<tangent<double>>;
    T xs[n], ys[n], res[n];

    // generate dummy data

    xs[0] = { .lb = { 0.5, 1.0 }, .ub = { 3.0, 1.0 } } ;

    ys[0] = { .lb = { 2.0, 0.0 }, .ub = { 5.0, 0.0 } };

    std::cout << xs[0] << std::endl;
    std::cout << ys[0] << std::endl;

    T *d_xs, *d_ys, *d_res;
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_ys, n * sizeof(*ys)));
    CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));

    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ys, ys, n * sizeof(*ys), hipMemcpyHostToDevice));

    kernel<<<n, 1>>>(d_xs, d_ys, d_res, n);

    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    auto r = res[0];
    std::cout << r << std::endl;

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_ys));
    CUDA_CHECK(hipFree(d_res));

    return 0;
}
