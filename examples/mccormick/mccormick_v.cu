#include "hip/hip_runtime.h"
#include "../common.h"

#include <hip/hip_runtime.h>

#include <cutangent/arithmetic/intrinsic_v.cuh>

#include <cumccormick/cumccormick.cuh>
#include <cumccormick/format.h>

#include <cutangent/cutangent.cuh>
#include <cutangent/format.h>

#include <cmath>
#include <iostream>

using cu::tangents;

#define PRINT_DEBUG 0

#define USE_VECTOR_LOAD_128 0
#define USE_VECTOR_LOAD_256 0

constexpr __device__ auto f(const auto &x, const auto &y, const auto &z, const auto &w)
{
    auto print = [](auto &x) {
#if 1 || PRINT_DEBUG
        printf("[gid:%3d][bid:%3d][tid:%3d] f {v {%g, %g, %g, %g}, cv [%g, %g, %g, %g] cc [%g, %g, %g, %g] lb [%g, %g, %g, %g] ub [%g, %g, %g, %g]}\n",
               threadIdx.x + blockIdx.x * blockDim.x, blockIdx.x, threadIdx.x,
               x.cv.v,
               x.cc.v,
               x.box.lb.v,
               x.box.ub.v,
               x.cv.ds[0], x.cv.ds[1], x.cv.ds[2], x.cv.ds[3],
               x.cc.ds[0], x.cc.ds[1], x.cc.ds[2], x.cc.ds[3],
               x.box.lb.ds[0], x.box.lb.ds[1], x.box.lb.ds[2], x.box.lb.ds[3],
               x.box.ub.ds[0], x.box.ub.ds[1], x.box.ub.ds[2], x.box.ub.ds[3]);
#endif
    };

    // printf("f {%g, [%g, %g], [%g, %g], [%g, %g]}\n", x.cv.v, x.cv.ds[0], x.cv.ds[1], y.cv.v, y.cv.ds[1], z.cv.v, z.cv.ds[2]);

    // auto a     = x * y + z + w;
    // auto a = x + y;
    // cu::mccormick<tangents<double, N>> a;

    print(x);
    // print(y);
    auto a = x + y + z + w;
    // auto b = a + a + a + a;
    // auto c = b + b + b;
    // auto d = c + c;
    // print(a);

    return a;
}

template<typename T, int N>
__global__ void
kernel(T *in, T *out, int n_elems, int n_vars)
{
    extern __shared__ cu::mccormick<cu::tangents<T, N>> xs[];

    int n                    = n_elems * n_vars;                      // total number of mccormick variables across all elements
    int gid                  = threadIdx.x + blockIdx.x * blockDim.x; // global id
    int bid                  = blockIdx.x;                            // block id
    int tid                  = threadIdx.x;                           // thread id inside block
    int n_threads            = blockDim.x;                            // number of threads in a block
    int n_blocks             = gridDim.x;                             // number of blocks in the grid  TODO: should probably be power of two for fast % operation
    int n_doubles_per_mc     = 4 * (N + 1);                           // 4 for cv, cc, lb, ub
    int n_out_doubles_per_mc = 2 * (N + 1);                           // 2 for cv, cc
    int xid                  = gid / n_vars;                          // mccormick id in xs

    // block range when considering only mccormick values (for initial copy from global memory)
    // int n_elems_per_block = (n_elems + n_blocks - 1) >> int(log2(n_blocks));
    int n_elems_per_block = (n_elems + n_blocks - 1) / n_blocks;
    int block_start       = n_elems_per_block * bid * 4 * n_vars;
    int block_end         = min(block_start + n_elems_per_block * 4 * n_vars, n * 4);

    // block range when considering tangents as well
    int n_elems_per_block_with_tangents = n_elems_per_block * n_vars * n_doubles_per_mc;
    int t_block_start                   = n_elems_per_block_with_tangents * bid;
    int t_block_end                     = min(t_block_start + n_elems_per_block_with_tangents, n_elems * n_vars * n_doubles_per_mc);

    if (tid == 0)
        printf("[gid:%3d][bid:%3d][tid:%3d][xid:%3d] elems_per_block: %3d block_start: %3d block_end: %3d\n",
               gid, bid, tid, xid, n_elems_per_block, block_start, block_end);

    // seed tangents
    // TODO: unroll
    for (int i = tid + t_block_start; i < t_block_end; i += n_threads) {
        int v = i / n_doubles_per_mc;
#if PRINT_DEBUG
        printf("[gid:%3d][bid:%3d][tid:%3d][vid:%3d][xid:%3d][i:%3d] n_elems_tangent: %3d t_block_start: %3d t_block_end: %3d\n",
               gid, bid, tid, v, xid, i, n_elems_per_block_with_tangents, t_block_start, t_block_end);
#endif

        int tangent_idx = (i % (N + 1)) - 1;                                    // tangent index for this thread, -1 is no tangent but a value to be skipped
                                                                                // faster alternative: int tangent_idx = x - floor(1/(N+1) * x) * (N+1) - 1; // TODO: check if compiler figures this out
        bool is_cv_or_cc                  = i % n_doubles_per_mc < 2 * (N + 1); // 2 since we only seed cv and cc
        ((double *)xs)[i - t_block_start] = (v % n_vars == tangent_idx) && is_cv_or_cc ? 1.0 : 0.0;

#if PRINT_DEBUG
        // printf("[gid:%3d][bid:%3d][tid:%3d][vid:%3d][tangent_idx:%3d][xid:%3d][i:%3d] tangent seed value: %g\n",
        //        gid, bid, tid, v, tangent_idx, xid, i - t_block_start, ((double *)xs)[i - t_block_start]);
#endif
    }

    __syncthreads();

    // Load elements from global memory into shared memory trying to get a balanced allocation in all blocks
#if USE_VECTOR_LOAD_128
    for (int i = tid * 2 + block_start; i + 1 < block_end; i += n_threads) {
        int sid                       = (i - block_start) * (N + 1);
        double2 tmp                   = *(double2 *)&in[i]; // init value
        ((double *)xs)[sid]           = tmp.x;
        ((double *)xs)[sid + (N + 1)] = tmp.y;
        // printf("[gid:%3d][bid:%3d][tid:%3d][i:%3d] init value: %g\n", gid, bid, tid, i, in[i]);
    }
#elif USE_VECTOR_LOAD_256
    for (int i = tid * 4 + block_start; i + 3 < block_end; i += n_threads) {
        int sid                           = (i - block_start) * (N + 1);
        double4 tmp                       = *(double4 *)&in[i]; // init value
        ((double *)xs)[sid]               = tmp.x;
        ((double *)xs)[sid + (N + 1)]     = tmp.y;
        ((double *)xs)[sid + 2 * (N + 1)] = tmp.z;
        ((double *)xs)[sid + 3 * (N + 1)] = tmp.w;
        // printf("[gid:%3d][bid:%3d][tid:%3d][i:%3d] init value: %g\n", gid, bid, tid, i, in[i]);
    }
#else
    for (int i = tid + block_start; i < block_end; i += n_threads) {
        int sid             = (i - block_start) * (N + 1);
        ((double *)xs)[sid] = in[i]; // init value
        // printf("[gid:%3d][bid:%3d][tid:%3d][i:%3d] init value: %g\n", gid, bid, tid, i, in[i]);
    }
#endif

    __syncthreads();

    // Actual computation
    int compute_out_offset  = n_elems_per_block * n_vars;
    int compute_block_start = n_elems_per_block * bid * N;
    int compute_block_end   = min(compute_block_start + n_elems_per_block * N, n_elems * N);
    // TODO: unroll
    for (int i = tid + compute_block_start; i < compute_block_end; i += n_threads) {
        // TODO: lid is wrong
        int lid = xid * n_vars - bid * n_threads; // local variable id
        // int lid = bid * n_elems_per_block; // local variable id
        int sid = (i - compute_block_start) % N;     // shared memory tangent id (subtracting compute_blocK_start is not really needed since we are starting at a multiple of N)
        int rid = compute_out_offset + lid / n_vars; // result id in shared memory - offset exists to not overwrite inputs that might be used for different sets of seed tangents
        // int rid = compute_out_offset + lid; // result id in shared memory - offset exists to not overwrite inputs that might be used for different sets of seed tangents

        // int vid = xid - bid * N;
        // int vid = gid / n_threads + bid * n_elems_per_block;

        // int lid = bid * n_elems_per_block + (tid / N) * n_vars; // local variable id
        // int sid = (i - compute_block_start);         // shared memory tangent id (subtracting compute_blocK_start is not really needed since we are starting at a multiple of N)
        int vid = bid * n_elems_per_block + tid / N;

        auto res = f(xs[lid], xs[lid + 1], xs[lid + 2], xs[lid + 7]); // TODO: add offset to vars for testing
        // auto res = f(xs[lid + 4], xs[lid + 5], xs[lid + 6], xs[lid + 7]); // TODO: add offset to vars for testing
        // auto res = f(xs[lid + 3], xs[lid + 4], xs[lid + 10], xs[lid + 12]); // TODO: add offset to vars for testing

        xs[rid].cv.ds[sid]     = res.cv.ds[sid];
        xs[rid].cc.ds[sid]     = res.cc.ds[sid];
        xs[rid].box.lb.ds[sid] = res.box.lb.ds[sid];
        xs[rid].box.ub.ds[sid] = res.box.ub.ds[sid];

        if (sid % N == 0) {
            // put res.v into shared memory
            xs[rid].cv.v     = res.cv.v;
            xs[rid].cc.v     = res.cc.v;
            xs[rid].box.lb.v = res.box.lb.v;
            xs[rid].box.ub.v = res.box.ub.v;
        }
        printf("A [gid:%3d][bid:%3d][tid:%3d][rid:%3d][vid:%3d][xid:%3d][lid:%3d][sid:%3d] in.v: %g %g %g %g res is: %g %g %g %g %g\n",
               gid, bid, tid, rid, vid, xid, lid, sid,
               xs[lid].cv.v, xs[lid + 1].cv.v, xs[lid + 2].cv.v, xs[lid + 7].cv.v,
               res.cv.v,
               res.cv.ds[sid], res.cc.ds[sid], res.box.lb.ds[sid], res.box.ub.ds[sid]);
        // TODO: maybe we can load it directly to global memory to save on shared memory space?
    }

    __syncthreads(); // TODO: probably not needed since the same thread is reading again? Make sure this is the case

    // Copy results from shared to global memory
    int out_sh_mem_offset = compute_out_offset * n_doubles_per_mc;
    int out_block_start   = n_elems_per_block * bid * n_out_doubles_per_mc;
    int out_block_end     = min(out_block_start + n_elems_per_block * n_out_doubles_per_mc, n_elems * n_out_doubles_per_mc);
    // TODO: unroll
    for (int i = tid + out_block_start; i < out_block_end; i += n_threads) {
        int sid = out_sh_mem_offset + i - out_block_start;
        out[i]  = ((double *)xs)[sid];
#if 0 || PRINT_DEBUG
        printf("[gid:%3d][bid:%3d][tid:%3d][bstart:%3d][bend:%3d] copy shared [%3d] (bank: [%3d]) into global [%3d] value: %g\n",
               gid, bid, tid, out_block_start, out_block_end, sid, sid % 32, i, out[i]);
#endif
    }

    //
    // Other tangent iterations (if n_tangents < n_vars)
    //

#if 0
    int tangent_offset     = N;
    int tangent_out_offset = n_elems * n_out_doubles_per_mc;

    int k = 1;
    // for (int k = 1; k < n_vars / N; k++) {
    //     printf("k is: %d\n", k);
    // seed tangents
    // TODO: unroll
    for (int i = tid + t_block_start; i < t_block_end; i += n_threads) {
        int v = i / n_doubles_per_mc;
#if PRINT_DEBUG
        printf("[gid:%3d][bid:%3d][tid:%3d][vid:%3d][xid:%3d][i:%3d][k:%d] n_elems_tangent: %3d t_block_start: %3d t_block_end: %3d\n",
               gid, bid, tid, v, xid, i, k, n_elems_per_block_with_tangents, t_block_start, t_block_end);
#endif

        int tangent_idx = (i % (N + 1)) + tangent_offset - 1; // tangent index for this thread, -1 is no tangent but a value to be skipped

        if (tangent_idx < 0)
            continue;

        bool is_cv_or_cc                  = (i + tangent_offset) % n_doubles_per_mc < 2 * (N + 1); // 2 since we only seed cv and cc
        ((double *)xs)[i - t_block_start] = (v % n_vars == tangent_idx) && is_cv_or_cc ? 1.0 : 0.0;

#if 0 || PRINT_DEBUG
        printf("[gid:%3d][bid:%3d][tid:%3d][vid:%3d][tangent_idx:%3d][xid:%3d][i:%3d][k:%d] tangent seed value: %g\n",
               gid, bid, tid, v, tangent_idx, xid, i - t_block_start, k, ((double *)xs)[i - t_block_start]);
#endif
    }

    __syncthreads();

    // Actual computation
    // TODO: unroll
    for (int i = tid + compute_block_start; i < compute_block_end; i += n_threads) {
        int lid = xid * n_vars - bid * n_threads;    // local variable id
        int sid = (i - compute_block_start) % N;     // shared memory tangent id (subtracting compute_blocK_start is not really needed since we are starting at a multiple of N)
        int rid = compute_out_offset + lid / n_vars; // result id in shared memory - offset exists to not overwrite inputs that might be used for different sets of seed tangents

        // auto res = f(xs[lid], xs[lid + 1], xs[lid + 2], xs[lid + 3]); // TODO: add offset to vars for testing
        // auto res = f(xs[lid + 8], xs[lid + 9], xs[lid + 10], xs[lid + 11]);
        auto res = f(xs[lid + 10], xs[lid + 11], xs[lid + 12], xs[lid + 13]);

        printf("[gid:%3d][bid:%3d][tid:%3d][rid:%3d] res is : %g %g %g %g %g\n", gid, bid, tid, rid, res.cv.v, res.cv.ds[sid], res.cc.ds[sid], res.box.lb.ds[sid], res.box.ub.ds[sid]);

        xs[rid].cv.ds[sid]     = res.cv.ds[sid];
        xs[rid].cc.ds[sid]     = res.cc.ds[sid];
        xs[rid].box.lb.ds[sid] = res.box.lb.ds[sid];
        xs[rid].box.ub.ds[sid] = res.box.ub.ds[sid];
    }

    // update inside
    // for (int i = tid + out_block_start; i < out_block_end; i += n_threads) {
    //     int sid      = out_sh_mem_offset + i - out_block_start;
    //     int out_idx  = i + tangent_offset;
    //     out[out_idx] = ((double *)xs)[sid];
    // }

    // update outside
    for (int i = tid + out_block_start; i < out_block_end; i += n_threads) {
        int sid      = out_sh_mem_offset + i - out_block_start;
        int out_idx  = i + tangent_out_offset;
        out[out_idx] = ((double *)xs)[sid]; // TODO: skip value in copy (already have it)

#if 1 || PRINT_DEBUG
        printf("[gid:%3d][bid:%3d][tid:%3d][bstart:%3d][bend:%3d][k:%d] copy shared [%3d] (bank: [%3d]) into global [%3d] value: %g\n",
               gid, bid, tid, out_block_start, out_block_end, k, sid, sid % 32, out_idx, out[out_idx]);
#endif
    }

#endif
    // }
}

/* we really have two scenarios where the GPU should be used differently.

1. n_tangents < n_vars -> multiple runs over the function to get to all tangents.

   a) if n_elems large and n_vars large, then we need to compute the tangents in a loop on the same threads.
      On different blocks/SMs we can compute the other elements in parallel.
   b) if n_elems small and n_vars small, then we can compute the different tangents in other blocks/SMs.

2. n_tangents == n_vars -> only parallelization of elements


we should first try to make use of all the blocks individually doing one mccormick computation (1a).
*/

int main()
{
    constexpr int n_elems               = 40;
    constexpr int n_vars                = 16;
    constexpr int n                     = n_elems * n_vars;
    constexpr int n_copy_doubles_per_mc = 2 * (n_vars + 1); // the number of doubles to copy from device back to host per mccormick relaxation. Skips box derivatives. Take cv, cc, lb, ub, cv.ds, cc.ds
    // constexpr int n_copy_doubles_per_mc = 4 + 4 * n_vars; // the number of doubles to copy from device back to host per mccormick relaxation. Skips box derivatives. Take cv, cc, lb, ub, cv.ds, cc.ds

    constexpr int n_blocks  = 10;
    constexpr int n_threads = 256;

    constexpr int n_tangents = 16; // the number of tangents to perform per mccormick relaxation, a multiple of 32 is ideal

    constexpr int n_elems_per_block = std::ceil(double(n_elems) / n_blocks);
    constexpr int n_vars_per_block  = n_vars * n_elems_per_block; // the number of mccormick variables to access in shared memory per block

    assert(n_mccormick >= n_vars && "n_mccormick must be >= n_vars");
    static_assert(n_tangents <= n_vars, "n_tangents must be <= n_vars");

    cu::mccormick<double> xs[n_elems * n_vars] {};
    double res[n_elems * n_copy_doubles_per_mc] {};

    // generate dummy data
    for (int i = 0; i < n_elems * n_vars; i += n_vars) {
        double v = i + 2;
        for (int j = 0; j < n_vars; j++) {
            xs[i + j] = v + j;
        }
    }

#if PRINT_DEBUG
    for (auto x : xs) {
        std::cout << "x is: " << x << std::endl;
    }
#endif

    constexpr int n_bytes_shared_in  = n_vars_per_block * 4 * sizeof(double) * (n_tangents + 1);
    constexpr int n_bytes_shared_out = n_elems_per_block * 4 * sizeof(double) * (n_tangents + 1);
    constexpr int n_bytes_shared     = n_bytes_shared_in + n_bytes_shared_out;
    printf("n_bytes_shared = %d B\n", n_bytes_shared);

    double *d_xs;  // we only use a single double array for easier coalescing
    double *d_res; // same as above
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_res, (n_elems * n_copy_doubles_per_mc) * sizeof(*res)));

    double *h_xs;
    CUDA_CHECK(hipHostMalloc(&h_xs, n * sizeof(*xs))); // 4 because of cv, cc, lb, ub
    memcpy(h_xs, xs, n * sizeof(*xs));

    CUDA_CHECK(hipMemcpy(d_xs, h_xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    kernel<double, n_tangents><<<n_blocks, n_threads, n_bytes_shared>>>(d_xs, d_res, n_elems, n_vars);
    CUDA_CHECK(hipMemcpy(res, d_res, (n_elems * n_copy_doubles_per_mc) * sizeof(*d_res), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

#if 1 || PRINT_DEBUG
    for (auto r : res) {
        std::cout << r << std::endl;
    }
#endif

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_res));

    CUDA_CHECK(hipHostFree(h_xs));

    return 0;
}
