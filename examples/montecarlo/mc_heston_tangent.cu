#include "hip/hip_runtime.h"
#include "../common.h"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cumccormick/cumccormick.cuh>
#include <cumccormick/format.h>

#include <cutangent/cutangent.cuh>
#include <cutangent/format.h>

#include <cstdint>
#include <iostream>

using u64 = std::uint64_t;

namespace cg = cooperative_groups;

thread_local double dummy = 0.0;

constexpr int N_THREADS = 512;

constexpr auto &value(auto &x) { return x; }
constexpr auto &derivative(auto &x) { return dummy; }

template<typename T>
constexpr auto european_call_payoff(const T &S, const T &K)
{
    using std::max;
    return max(S - K, T {});
};

namespace heston
{

template<typename T>
struct parameters
{
    T r;     // risk-free interest rate
    T S0;    // spot price
    T tau;   // time until maturity
    T K;     // strike price
    T v0;    // initial volatility
    T rho;   // correlation of asset and volatility
    T kappa; // mean-reversion rate
    T theta; // long run average volatility
    T xi;    // volatility of volatility
};

template<typename T>
struct state
{
    T S_t; // current asset price
    T v_t; // current volatility
};

template<typename T>
constexpr state<T> step(state<T> &state, const auto &Z_t, const auto &dt, const parameters<T> &params)
{
    auto [r, S0, tau, K, v0, rho, kappa, theta, xi] = params;
    auto [S_t, v_t]                                 = state;

    using std::abs;
    using std::exp;
    using std::max;
    using std::pow;
    using std::sqrt;

    constexpr T zero {};

    v_t = max(v_t, zero); // full truncation

    // using the Euler-Maryuama discretization scheme
    return { .S_t = S_t * exp((r - 0.5 * v_t) * dt + sqrt(v_t * dt) * Z_t.y),
             .v_t = v_t + kappa * (theta - v_t) * dt + xi * sqrt(v_t * dt) * Z_t.x };
}

}; // namespace heston

template<typename T>
__device__ void reduce_block(T *sum, cg::thread_block &cta, cg::thread_block_tile<32> &tile32, T *res)
{
    const int VEC = 32;
    const int tid = cta.thread_rank();

    T beta = sum[tid];
    T temp;

    // reduction per warp
    for (int i = VEC / 2; i > 0; i >>= 1) {
        if (tile32.thread_rank() < i) {
            temp = sum[tid + i];
            beta += temp;
            sum[tid] = beta;
        }
        cg::sync(tile32);
    }
    cg::sync(cta);

    if (tid == 0) {
        beta = 0;
        for (int i = 0; i < blockDim.x; i += VEC) {
            beta += sum[i];
        }
        *res += beta;
    }
    cg::sync(cta);
}

namespace monte_carlo
{
struct parameters
{
    u64 n_options; // how many option scenarios to calculate
    u64 n_paths;   // how many paths to take per monte carlo simulation
    u64 n_steps;   // how many steps to take per path
};
} // namespace monte_carlo

template<typename T>
__global__ void heston_monte_carlo(monte_carlo::parameters mc_params,
                                   hiprandState *rng_states,
                                   heston::parameters<T> *ps,
                                   T *tmp,
                                   T *res)
{
    using std::exp;
    using std::pow;
    using std::sqrt;

    const auto [n_options, n_paths, n_steps] = mc_params;

    cg::thread_block cta             = cg::this_thread_block();
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;

    const int n_blocks  = gridDim.x;
    const int n_threads = blockDim.x;

    const u64 n_paths_per_block = n_paths / n_blocks;

    __shared__ T payoffs[N_THREADS];

    if (tid < n_threads) {
        payoffs[tid] = 0.0;
    }
    cg::sync(cta);

    hiprandState rng_state = rng_states[gid];

    int i = 0;

    const auto [r, S0, tau, K, v0, rho_, kappa, theta, xi] = ps[i];

    const auto rho = value(rho_);
    const auto dt  = tau / n_steps;

    T accum = 0.0;

    for (int j = tid; j < n_paths_per_block; j += n_threads) {
        heston::state<T> state { .S_t = S0, .v_t = v0 };
        double2 Z_t;

        for (int k = 0; k < n_steps; k++) {
            Z_t = hiprand_normal2_double(&rng_state);

            // correlate the two random numbers
            Z_t.y = rho * Z_t.x + sqrt(1.0 - pow(rho, 2)) * Z_t.y;

            state = heston::step(state, Z_t, dt, ps[i]);
        }

        auto payoff  = european_call_payoff(state.S_t, K);
        payoffs[tid] = payoff;

        cg::sync(cta);

        reduce_block<T>(payoffs, cta, tile32, &accum);
    }

    if (tid == 0) {
        tmp[bid] = accum;
    }

    rng_states[gid] = rng_state;
}

template<typename T>
__global__ void heston_price_from_payoffs(monte_carlo::parameters mc_params,
                                          hiprandState *rng_states,
                                          heston::parameters<T> *ps,
                                          T *tmp,
                                          T *res)
{
    const auto [n_options, n_paths, n_steps] = mc_params;

    cg::thread_block cta             = cg::this_thread_block();
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    int i = 0; // we currently evaluate only one option

    const auto [r, S0, tau, K, v0, rho_, kappa, theta, xi] = ps[i];

    if (bid == 0) {
        T final_payoff_sum = 0.0;
        reduce_block<T>(tmp, cta, tile32, &final_payoff_sum);

        if (tid == 0) {
            T call_price = (final_payoff_sum / n_paths) * exp(-r * tau);

            res[i] = call_price;
        }
    }
}

__global__ void rng_init(auto *rng_states)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(bid, tid, 0, &rng_states[gid]); // each block gets a different seed
}

int main()
{
    constexpr int n         = 1;
    constexpr int n_threads = N_THREADS;
    constexpr int n_blocks  = 1024;

    using T = cu::tangent<double>;
    heston::parameters<T> xs[n] {};
    T res[n];

    // generate dummy scenarios
    for (int i = 0; i < n; i++) {
        value(xs[i].r)     = 0.0319;
        value(xs[i].S0)    = 100.0;
        value(xs[i].tau)   = 1.0;
        value(xs[i].K)     = 100.0;
        value(xs[i].v0)    = 0.010201;
        value(xs[i].rho)   = -0.7;
        value(xs[i].kappa) = 6.21;
        value(xs[i].theta) = 0.019;
        value(xs[i].xi)    = 0.61;

        // update seeds to compute derivative w.r.t S0
        derivative(xs[i].S0) = 1.0;
    }

    monte_carlo::parameters mc_params { .n_options = n,
                                        .n_paths   = 1024 * 1024,
                                        .n_steps   = 1024 };

    std::cout << "---- Computing Delta ----" << std::endl;
    std::cout << "S0: " << xs[0].S0 << std::endl;
    std::cout << "v0: " << xs[0].v0 << std::endl;

    heston::parameters<T> *d_xs;
    T *d_res;
    T *d_tmp;
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));
    CUDA_CHECK(hipMalloc(&d_tmp, n_blocks * sizeof(*d_tmp)));

    hiprandState *rng_states;
    CUDA_CHECK(hipMalloc((void **)&rng_states, n_blocks * n_threads * sizeof(hiprandState)));
    CUDA_CHECK(hipMemset(rng_states, 0, n_blocks * n_threads * sizeof(hiprandState)));
    rng_init<<<n_blocks, n_threads>>>(rng_states);

    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    heston_monte_carlo<<<n_blocks, n_threads>>>(mc_params, rng_states, d_xs, d_tmp, d_res);
    heston_price_from_payoffs<<<1, n_blocks>>>(mc_params, rng_states, d_xs, d_tmp, d_res);
    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    for (auto value_and_delta : res) {
        std::cout << "Heston European call w.r.t. S0 (i.e., Delta): " << value_and_delta << std::endl;
    }

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_res));

    return 0;
}
